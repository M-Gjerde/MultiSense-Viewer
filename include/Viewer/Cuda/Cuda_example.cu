#include "hip/hip_runtime.h"
//
// Created by mgjer on 15/01/2024.
//

#define NOMINMAX
#define GLFW_INCLUDE_VULKAN
#ifdef _WIN64
#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>
#define _USE_MATH_DEFINES
#endif

#include <vulkan/vulkan.h>
#include <GLFW/glfw3.h>
#ifdef _WIN64
#include <vulkan/vulkan_win32.h>
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Viewer/Tools/hip/hip_runtime_api.h>

#include <rasterize_points.h>
#include <opencv2/opencv.hpp>
#include <tinyply.h>

#include <Viewer/Tools/hip/hip_runtime_api.h>
#include "Viewer/Cuda/Cuda_example.h"

#include <glm/trigonometric.hpp>

#include "Viewer/Core/Texture.h"
#include "Viewer/Tools/Logger.h"

void printTensor(const torch::Tensor& tensor, int64_t numRows = 1) {
    std::cout << "Shape/Size: " << tensor.sizes() << std::endl;
    std::cout << "Data Type: " << tensor.dtype() << std::endl;
    std::cout << "Device: " << tensor.device() << std::endl;
    if (numRows > 0) {
        int rowsToPrint = std::min(numRows, tensor.size(0));
        for (int i = 0; i < rowsToPrint; ++i) {
            // Print each row. Assuming tensor is 2D.
            for (int j = 0; j < tensor.size(1); ++j) {
                std::cout << tensor[i][j].item<float>() << " ";
            }
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
}

inline std::vector<uint8_t> read_file_binary(const std::string& pathToFile) {
    std::ifstream file(pathToFile, std::ios::binary);
    std::vector<uint8_t> fileBufferBytes;

    if (file.is_open()) {
        file.seekg(0, std::ios::end);
        size_t sizeBytes = file.tellg();
        file.seekg(0, std::ios::beg);
        fileBufferBytes.resize(sizeBytes);
        if (file.read((char*)fileBufferBytes.data(), sizeBytes)) return fileBufferBytes;
    }
    else throw std::runtime_error("could not open binary ifstream to path " + pathToFile);
    return fileBufferBytes;
}

struct memory_buffer : public std::streambuf {
    char* p_start{nullptr};
    char* p_end{nullptr};
    size_t size;

    memory_buffer(char const* first_elem, size_t size)
        : p_start(const_cast<char*>(first_elem)), p_end(p_start + size), size(size) {
        setg(p_start, p_start, p_end);
    }

    pos_type seekoff(off_type off, std::ios_base::seekdir dir, std::ios_base::openmode which) override {
        if (dir == std::ios_base::cur) gbump(static_cast<int>(off));
        else setg(p_start, (dir == std::ios_base::beg ? p_start : p_end) + off, p_end);
        return gptr() - p_start;
    }

    pos_type seekpos(pos_type pos, std::ios_base::openmode which) override {
        return seekoff(pos, std::ios_base::beg, which);
    }
};

struct memory_stream : virtual memory_buffer, public std::istream {
    memory_stream(char const* first_elem, size_t size)
        : memory_buffer(first_elem, size), std::istream(static_cast<std::streambuf*>(this)) {
    }
};


// GaussianData class definition
class GaussianData {
public:
    torch::Tensor xyz;
    torch::Tensor rot;
    torch::Tensor scale;
    torch::Tensor opacity;
    torch::Tensor sh;

    torch::Tensor flat() const {
        return torch::cat({xyz, rot, scale, opacity, sh}, -1).contiguous();
    }

    int64_t length() const {
        return xyz.size(0);
    }

    int sh_dim() const {
        return sh.size(-1);
    }
};

torch::Tensor ConvertGlmMat4ToTensor(const glm::mat4& mat) {
    // Create a 4x4 tensor
    torch::Tensor tensor = torch::empty({4, 4}, torch::kFloat32);

    // Copy data from glm::mat4 to the tensor
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            tensor[i][j] = mat[i][j];
        }
    }
    return tensor;
}

torch::Tensor ConvertGlmVec3ToTensor(const glm::vec3& vec) {
    // Create a 1D tensor with 3 elements
    torch::Tensor tensor = torch::empty({3}, torch::kFloat32);

    // Copy data from glm::vec3 to the tensor
    tensor[0] = vec.x;
    tensor[1] = vec.y;
    tensor[2] = vec.z;

    return tensor;
}

GaussianData naive_gaussian() {
    auto gau_xyz = torch::tensor({
                                     0, 0, 0,
                                     1, 0, 0,
                                     0, 1, 0,
                                     0, 0, 1
                                 },
                                 torch::dtype(torch::kFloat32)).view({-1, 3});

    auto gau_rot = torch::tensor({
                                     1, 0, 0, 0,
                                     1, 0, 0, 0,
                                     1, 0, 0, 0,
                                     1, 0, 0, 0
                                 },
                                 torch::dtype(torch::kFloat32)).view({-1, 4});
    auto gau_s = torch::tensor({
                                   0.1, 0.1, 0.1,
                                   0.2, 0.03, 0.03,
                                   0.03, 0.2, 0.03,
                                   0.03, 0.03, 0.2
                               },
                               torch::dtype(torch::kFloat32)).view({-1, 3});
    auto gau_c = torch::tensor({
                                   1, 0, 1,
                                   1, 0, 0,
                                   0, 1, 0,
                                   0, 0, 1,
                               },
                               torch::dtype(torch::kFloat32)).view({-1, 3});
    gau_c = (gau_c - 0.5) / 0.28209;

    auto gau_a = torch::tensor({1, 1, 1, 1}, torch::dtype(torch::kFloat32)).view({-1, 1});

    return GaussianData(gau_xyz, gau_rot, gau_s, gau_a, gau_c);
}

GaussianData loadTinyPly(std::filesystem::path filePath, bool preloadIntoMemory = true) {
    std::cout << "........................................................................\n";
    std::cout << "Now Reading: " << filePath << std::endl;

    std::unique_ptr<std::istream> file_stream;
    std::vector<uint8_t> byte_buffer;
    torch::Tensor xyzTensor, rotationsTensor, scalesTensor, opacitiesTensor, shsTensor, reshapedTensor;
    try {
        // For most files < 1gb, pre-loading the entire file upfront and wrapping it into a
        // stream is a net win for parsing speed, about 40% faster.
        if (preloadIntoMemory) {
            byte_buffer = read_file_binary(filePath.string());
            file_stream.reset(new memory_stream((char*)byte_buffer.data(), byte_buffer.size()));
        }

        if (!file_stream || file_stream->fail())
            throw std::runtime_error(
                "file_stream failed to open " + filePath.string());

        file_stream->seekg(0, std::ios::end);
        const float size_mb = file_stream->tellg() * float(1e-6);
        file_stream->seekg(0, std::ios::beg);

        tinyply::PlyFile file;
        file.parse_header(*file_stream);

        std::cout << "\t[ply_header] Type: " << (file.is_binary_file() ? "binary" : "ascii") << std::endl;
        for (const auto& c : file.get_comments()) std::cout << "\t[ply_header] Comment: " << c << std::endl;
        for (const auto& c : file.get_info()) std::cout << "\t[ply_header] Info: " << c << std::endl;

        for (const auto& e : file.get_elements()) {
            std::cout << "\t[ply_header] element: " << e.name << " (" << e.size << ")" << std::endl;
            for (const auto& p : e.properties) {
                std::cout << "\t[ply_header] \tproperty: " << p.name << " (type=" << tinyply::PropertyTable[p.
                    propertyType].str << ")";
                if (p.isList) std::cout << " (list_type=" << tinyply::PropertyTable[p.listType].str << ")";
                std::cout << std::endl;
            }
        }

        // Because most people have their own mesh types, tinyply treats parsed data as structured/typed byte buffers.
        // See examples below on how to marry your own application-specific data structures with this one.
        std::shared_ptr<tinyply::PlyData> vertices, normals, f_dc, f_rest, opacities, scales, rotations;

        // The header information can be used to programmatically extract properties on elements
        // known to exist in the header prior to reading the data. For brevity of this sample, properties
        // like vertex position are hard-coded:
        try { vertices = file.request_properties_from_element("vertex", {"x", "y", "z"}); }
        catch (const std::exception& e) { std::cerr << "tinyply exception: " << e.what() << std::endl; }

        try { normals = file.request_properties_from_element("vertex", {"nx", "ny", "nz"}); }
        catch (const std::exception& e) { std::cerr << "tinyply exception: " << e.what() << std::endl; }

        try { f_dc = file.request_properties_from_element("vertex", {"f_dc_0", "f_dc_1", "f_dc_2"}); }
        catch (const std::exception& e) { std::cerr << "tinyply exception: " << e.what() << std::endl; }

        std::vector<std::string> propertyKeys;
        for (size_t i = 0; i < 45; ++i) {
            propertyKeys.push_back("f_rest_" + std::to_string(i));;
        }
        try { f_rest = file.request_properties_from_element("vertex", propertyKeys); }
        catch (const std::exception& e) { std::cerr << "tinyply exception: " << e.what() << std::endl; }

        try { opacities = file.request_properties_from_element("vertex", {"opacity"}); }
        catch (const std::exception& e) { std::cerr << "tinyply exception: " << e.what() << std::endl; }

        // Providing a list size hint (the last argument) is a 2x performance improvement. If you have
        // arbitrary ply files, it is best to leave this 0.
        try { scales = file.request_properties_from_element("vertex", {"scale_0", "scale_1", "scale_2"}); }
        catch (const std::exception& e) { std::cerr << "tinyply exception: " << e.what() << std::endl; }

        // Tristrips must always be read with a 0 list size hint (unless you know exactly how many elements
        // are specifically in the file, which is unlikely);
        try { rotations = file.request_properties_from_element("vertex", {"rot_0", "rot_1", "rot_2", "rot_3"}, 0); }
        catch (const std::exception& e) { std::cerr << "tinyply exception: " << e.what() << std::endl; }


        if (vertices) std::cout << "\tRead " << vertices->count << " total vertices " << std::endl;
        if (normals) std::cout << "\tRead " << normals->count << " total vertex normals " << std::endl;
        if (f_dc) std::cout << "\tRead " << f_dc->count << " total vertex f_dc " << std::endl;
        if (f_rest) std::cout << "\tRead " << f_rest->count << " total vertex f_rest " << std::endl;
        if (opacities) std::cout << "\tRead " << opacities->count << " total vertex opacities " << std::endl;
        if (scales) std::cout << "\tRead " << scales->count << " total scales " << std::endl;
        if (rotations) std::cout << "\tRead " << rotations->count << " total rotations) " << std::endl;

        try {
            file.read(*file_stream);
        }
        catch (const std::exception& e) {
            std::cerr << "tinyply exception (reading data): " << e.what() << std::endl;
        }

        // Example One: converting to your own application types
        {
            const size_t numVerticesBytes = vertices->buffer.size_bytes();
            if (vertices->t != tinyply::Type::FLOAT32) {
                std::cerr << "Data type is not float32" << std::endl;
            }
            int64_t count = vertices->count;
            xyzTensor = torch::zeros({count, 3}, torch::kFloat32);
            std::memcpy(xyzTensor.data_ptr(), vertices->buffer.get(), numVerticesBytes);

            rotationsTensor = torch::zeros({static_cast<int64_t>(rotations->count), 4}, torch::kFloat32);
            std::memcpy(rotationsTensor.data_ptr(), rotations->buffer.get(), rotations->buffer.size_bytes());
            torch::Tensor norms = rotationsTensor.norm(2, -1, true);
            printTensor(rotationsTensor, 10);
            rotationsTensor = rotationsTensor / norms;
            printTensor(rotationsTensor, 10);

            scalesTensor = torch::zeros({static_cast<int64_t>(scales->count), 3}, torch::kFloat32);
            std::memcpy(scalesTensor.data_ptr(), scales->buffer.get(), scales->buffer.size_bytes());

            opacitiesTensor = torch::zeros({static_cast<int64_t>(opacities->count), 1}, torch::kFloat32);
            std::memcpy(opacitiesTensor.data_ptr(), opacities->buffer.get(), opacities->buffer.size_bytes());
            printTensor(opacitiesTensor, 5);
            opacitiesTensor = torch::sigmoid(opacitiesTensor);
            printTensor(opacitiesTensor, 5);

            torch::Tensor dcFeatures = torch::zeros({static_cast<int64_t>(f_dc->count), 3}, torch::kFloat32);
            std::memcpy(dcFeatures.data_ptr(), f_dc->buffer.get(), f_dc->buffer.size_bytes());

            int maxShDegree = 3;
            torch::Tensor extraFeatures = torch::zeros({static_cast<int64_t>(f_rest->count), 45}, torch::kFloat32);
            std::memcpy(extraFeatures.data_ptr(), f_rest->buffer.get(), f_rest->buffer.size_bytes());
            extraFeatures = extraFeatures.reshape({
                extraFeatures.size(0), 3, static_cast<int64_t>((maxShDegree + 1) * (maxShDegree + 1) - 1)
            });
            extraFeatures = extraFeatures.transpose(1, 2);
            dcFeatures = dcFeatures.unsqueeze(1);
            dcFeatures = dcFeatures.transpose(1, 2);

            std::cout << "dcFeatures shape: " << dcFeatures.sizes() << std::endl;
            std::cout << "extraFeatures shape: " << extraFeatures.sizes() << std::endl;

            shsTensor = torch::concatenate({
                                               dcFeatures.reshape({-1, 3}),
                                               extraFeatures.reshape({dcFeatures.size(0), -1})
                                           }, -1);
            xyzTensor *= 10;

            printTensor(xyzTensor);
        }
    }
    catch (const std::exception& e) {
        std::cerr << "Caught tinyply exception: " << e.what() << std::endl;
    }


    return {xyzTensor, rotationsTensor, scalesTensor, opacitiesTensor, shsTensor};
}


CudaImplementation::CudaImplementation(VkInstance* instance, VkDevice device, const RasterSettings* settings,
                                       const std::filesystem::path& modelPath, uint32_t memSizeCuda,
                                       std::vector<TextureCuda>* textures
) {
    uint32_t type = sizeof(uint8_t);
    uint32_t channels = 4;
    uint32_t imageSize = settings->imageHeight * settings->imageWidth * channels * type;
    cudaExtMem.resize(textures->size());
    cudaMemPtr.resize(textures->size());
    cudaMipMappedArrays.resize(textures->size());
    cudaFirstLevels.resize(textures->size());

    PFN_vkGetMemoryWin32HandleKHR fpGetMemoryWin32HandleKHR = reinterpret_cast<PFN_vkGetMemoryWin32HandleKHR>(
        vkGetInstanceProcAddr(*instance, "vkGetMemoryWin32HandleKHR"));
    if (fpGetMemoryWin32HandleKHR == nullptr) {
        Log::Logger::getInstance()->error("Function not available");
    }

    for (size_t i = 0; i < cudaExtMem.size(); ++i) {
        void* handle;
        VkMemoryGetWin32HandleInfoKHR vkMemoryGetWin32HandleInfoKHR = {};
        vkMemoryGetWin32HandleInfoKHR.sType =
            VK_STRUCTURE_TYPE_MEMORY_GET_WIN32_HANDLE_INFO_KHR;
        vkMemoryGetWin32HandleInfoKHR.pNext = NULL;
        vkMemoryGetWin32HandleInfoKHR.memory = (*textures)[i].m_DeviceMemory;
        vkMemoryGetWin32HandleInfoKHR.handleType = VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT;

        if (fpGetMemoryWin32HandleKHR(device, &vkMemoryGetWin32HandleInfoKHR,
                                      &handle) !=
            VK_SUCCESS) {
            Log::Logger::getInstance()->error("vkGetMemoryWin32HandleKHR not available");
        }

        hipExternalMemoryHandleDesc cudaExtMemHandleDesc{};
        memset(&cudaExtMemHandleDesc, 0, sizeof(cudaExtMemHandleDesc));
        cudaExtMemHandleDesc.size = memSizeCuda;
        cudaExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
        cudaExtMemHandleDesc.handle.win32.handle = handle;
        cudaExtMemHandleDesc.flags = 0;
        checkCudaErrors(hipImportExternalMemory(&cudaExtMem[i], &cudaExtMemHandleDesc));


        hipChannelFormatDesc formatDesc;
        memset(&formatDesc, 0, sizeof(formatDesc));
        formatDesc.x = 8;
        formatDesc.y = 8;
        formatDesc.z = 8;
        formatDesc.w = 8;
        formatDesc.f = hipChannelFormatKindUnsigned;

        hipExtent extent = make_hipExtent(settings->imageWidth, settings->imageHeight, 0);

        unsigned int flags = 0;
        //flags |= cudaArrayColorAttachment;

        cudaExternalMemoryMipmappedArrayDesc desc = {};
        memset(&desc, 0, sizeof(desc));
        desc.offset = 0;
        desc.formatDesc = formatDesc;
        desc.extent = extent;
        desc.flags = flags;
        desc.numLevels = 1;

        checkCudaErrors(cudaExternalMemoryGetMappedMipmappedArray(&cudaMipMappedArrays[i], cudaExtMem[i], &desc));

        checkCudaErrors(hipGetMipmappedArrayLevel(&cudaFirstLevels[i], cudaMipMappedArrays[i], 0));
        // 0 for the first level
    }


    torch::Device cudaDevice(torch::kCUDA);

    campos = ConvertGlmVec3ToTensor(settings->camPos).to(cudaDevice);
    bg = torch::tensor({0.0, 0.0, 0.0}, torch::dtype(torch::kFloat32)).to(cudaDevice);
    // Other parameters
    scale_modifier = settings->scaleModifier;
    tan_fovx = settings->tanFovX;
    tan_fovy = settings->tanFovY;
    image_height = settings->imageHeight;
    image_width = settings->imageWidth;
    degree = settings->shDegree;
    prefiltered = settings->prefilter;
    debug = settings->debug;
    std::filesystem::path pointCloudPath = "point_cloud/iteration_7000/point_cloud.ply";
    std::filesystem::path fullPath = modelPath / pointCloudPath;
    auto gaussianData = loadTinyPly(fullPath);
    //auto gaussianData = naive_gaussian();
    // Example usage
    if (!gaussianData.length()) {
        gaussianData = naive_gaussian();
    }
    means3D = gaussianData.xyz.to(cudaDevice) * 100;
    shs = gaussianData.sh.to(cudaDevice);
    opacity = gaussianData.opacity.to(cudaDevice);
    scales = gaussianData.scale.to(cudaDevice);
    rotations = gaussianData.rot.to(cudaDevice);
    cov3Dprecompute = torch::tensor({}).to(cudaDevice);
    colors = torch::tensor({}).to(cudaDevice);
    degree = 3;

    shs = shs.view({gaussianData.length(), -1, 3}).contiguous();
}

void CudaImplementation::updateGaussianData() {
}


void CudaImplementation::updateCameraPose(glm::mat4 view, glm::mat4 proj, glm::vec3 target) {
    torch::Device device(torch::kCUDA);
    glm::vec3 cameraPos = glm::inverse(view)[3];
    /*
    glm::vec3 cameraPos = glm::inverse(view)[3];
    //glm::vec3 cameraPos(3.0f, -3.0f, 0.0f);
    glm::vec3 dirNorm = glm::normalize(target - cameraPos);
    glm::vec3 worldUp(0.0f, 0.0f, 1.0f);
    glm::vec3 right = glm::cross(worldUp, dirNorm);
    glm::vec3 cameraUp = glm::cross(dirNorm, right);
    glm::mat4 cameraTrans(
        right.x, right.y, right.z, 0,
        cameraUp.x, cameraUp.y, cameraUp.z, 0,
        -dirNorm.x, -dirNorm.y, -dirNorm.z, 0,
        cameraPos.x, cameraPos.y, cameraPos.z, 1
    );
    view = glm::inverse(cameraTrans);
    //std::cout << dirNorm.x << ", " << dirNorm.y << ", " << dirNorm.z << std::endl;

    //view[0][0] *= -1;
    //view[1][0] *= -1;
    //view[2][0] *= -1;
    //view[3][0] *= -1;
    ////
    //view[0][2] *= -1;
    //view[1][2] *= -1;
    //view[2][2] *= -1;
    //view[3][2] *= -1;

    float aspect = 1280.0f / 720.0f;
    float farPlane = 100;
    float nearPlane = 0.01;
    float focal_length = 1.0f / tan(glm::radians(60.0f) * 0.5f);
    float x = focal_length / aspect;
    float y = -focal_length;
    float A = -farPlane / (farPlane - nearPlane);
    float B = -farPlane * nearPlane / (farPlane - nearPlane);


    proj = glm::mat4(
        x, 0.0f, 0.0f, 0.0f,
        0.0f, y, 0.0f, 0.0f,
        0.0f, 0.0f, A, -1.0f,
        0.0f, 0.0f, B, 0.0f);
    */

    glm::mat4 projView = proj * view;
    viewmatrix = ConvertGlmMat4ToTensor(view).to(device);
    projmatrix = ConvertGlmMat4ToTensor(projView).to(device);
    //printTensor(viewmatrix, true);
    //printTensor(projmatrix, true);
    campos = ConvertGlmVec3ToTensor(cameraPos).to(torch::kFloat).to(device);
}

void CudaImplementation::updateSettings(const CudaImplementation::RasterSettings& settings) {
    scale_modifier = settings.scaleModifier;
}

void CudaImplementation::updateCameraIntrinsics(float hfox, float hfovy) {
    tan_fovx = hfox;
    tan_fovy = hfovy;
}


void CudaImplementation::draw(uint32_t i, void* streamToRun) {
    int rendered;
    torch::Tensor out_color, radii, geomBuffer, binningBuffer, imgBuffer;
    // Call the function
    try {
        std::tie(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer) = RasterizeGaussiansCUDA(
            bg, means3D, colors, opacity, scales, rotations,
            scale_modifier, cov3Dprecompute, viewmatrix, projmatrix, tan_fovx, tan_fovy,
            image_height, image_width, shs, degree, campos, prefiltered, debug, streamToRun
        );
    }
    catch (const std::exception& e) {
        std::cerr << "Caught exception: " << e.what() << std::endl;
        rendered = 0;
    }
    // Ensure the tensor is on the CPU and is a byte tensor
    if (rendered == 0) {
        return;
    }
    auto img = out_color.permute({1, 2, 0}); // Change [Channels, Height, Width] to [Height, Width, Channels]
    img = img.contiguous();
    auto alpha_channel = torch::ones({img.size(0), img.size(1), 1}, img.options());
    auto img_with_alpha = torch::cat({img, alpha_channel}, 2);
    img_with_alpha = img_with_alpha.contiguous();
    img_with_alpha *= 255;
    img_with_alpha = img_with_alpha.to(torch::kU8);
    auto img_with_alpha_ptr = img_with_alpha.data_ptr<uint8_t>();
    hipMemcpy3DParms p{};
    memset(&p, 0x00, sizeof(hipMemcpy3DParms));

    p.srcPtr = make_hipPitchedPtr(img_with_alpha_ptr, image_width * 4, image_width, image_height);
    p.dstArray = cudaFirstLevels[i];
    p.extent = make_hipExtent(image_width, image_height, 1); // depth is 1 for 2D
    p.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&p));
    /*
    try {
        if (img.device().is_cuda()) {
            img = img.to(torch::kCPU);
            // Make sure the tensor is contiguous and in the format [Height, Width, Channels]
            img = img.contiguous();
            //img = img * 255;
            //torch::Tensor img_uchar = img.to(torch::kU8);
            //printTensor(img_uchar, 0);

            auto channels = CV_32FC(img.size(2));
            cv::Mat mat(img.size(0), img.size(1), channels,
                        img.data_ptr<float>());
            cv::Mat img_flipped;
            cv::Mat img_flipped_x;
            cv::flip(mat, img_flipped, 0); // Flip the image vertically
            cv::cvtColor(img_flipped, img_flipped, cv::COLOR_BGR2RGB);
            // Display the image
            cv::imshow("Output Image", img_flipped);
            //cv::imshow("Output Image flipped x", img_flipped_x);
            cv::waitKey(1); // Wait for a key press (use 0 for infinite wait)
        }
    }
    catch (const torch::Error& e) {
        std::cerr << "Error during tensor device check or transfer: " << e.what() << std::endl;
    }
    */

    //checkCudaErrors(hipMemcpy(cudaMipMappedArrays[i],img_with_alpha.data_ptr() , data_size, hipMemcpyDeviceToDevice));
}
