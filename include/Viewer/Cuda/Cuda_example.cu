#include "hip/hip_runtime.h"
//
// Created by mgjer on 15/01/2024.
//

#include <iostream>
#include <random>
#include <rasterize_points.h>

#include "Viewer/Cuda/Cuda_example.h"

#include <glm/ext/quaternion_transform.hpp>
#include <glm/ext/quaternion_trigonometric.hpp>

#include "Viewer/Tools/Logger.h"
#include <Viewer/Core/Texture.h>

#include <torch/torch.h>
#include <Viewer/Tools/hip/hip_runtime_api.h>

#include <opencv2/opencv.hpp>
#include <glm/gtx/transform.hpp>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

void printTensor(const torch::Tensor& tensor, bool printContents = false) {
    std::cout << "Shape/Size: " << tensor.sizes() << std::endl;
    std::cout << "Data Type: " << tensor.dtype() << std::endl;
    std::cout << "Device: " << tensor.device() << std::endl;
    if (printContents)
        std::cout << "Data: " << tensor << std::endl;
}

// GaussianData class definition
class GaussianData {
public:
    torch::Tensor xyz;
    torch::Tensor rot;
    torch::Tensor scale;
    torch::Tensor opacity;
    torch::Tensor sh;

    torch::Tensor flat() const {
        return torch::cat({xyz, rot, scale, opacity, sh}, -1).contiguous();
    }

    size_t length() const {
        return xyz.size(0);
    }

    int sh_dim() const {
        return sh.size(-1);
    }
};

struct PlyVertex {
    float x, y, z; // Position
    float nx, ny, nz; // Normal
    float f_dc[3]; // f_dc properties
    float f_rest[44]; // f_rest properties
    float opacity;
    float scale[3];
    float rot[4];
};

torch::Tensor ConvertGlmMat4ToTensor(const glm::mat4& mat) {
    // Create a 4x4 tensor
    torch::Tensor tensor = torch::empty({4, 4}, torch::kFloat32);

    // Copy data from glm::mat4 to the tensor
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            tensor[i][j] = mat[i][j];
        }
    }
    return tensor;
}

torch::Tensor ConvertGlmVec3ToTensor(const glm::vec3& vec) {
    // Create a 1D tensor with 3 elements
    torch::Tensor tensor = torch::empty({3}, torch::kFloat32);

    // Copy data from glm::vec3 to the tensor
    tensor[0] = vec.x;
    tensor[1] = vec.y;
    tensor[2] = vec.z;

    return tensor;
}

GaussianData naive_gaussian() {
    auto gau_xyz = torch::tensor({
                                     0, 0, 0,
                                     1, 0, 0,
                                     0, 1, 0,
                                     0, 0, 1
                                 },
                                 torch::dtype(torch::kFloat32)).view({-1, 3});

    auto gau_rot = torch::tensor({
                                     1, 0, 0, 0,
                                     1, 0, 0, 0,
                                     1, 0, 0, 0,
                                     1, 0, 0, 0
                                 },
                                 torch::dtype(torch::kFloat32)).view({-1, 4});
    auto gau_s = torch::tensor({
                                   0.03, 0.03, 0.03,
                                   0.2, 0.03, 0.03,
                                   0.03, 0.2, 0.03,
                                   0.03, 0.03, 0.2
                               },
                               torch::dtype(torch::kFloat32)).view({-1, 3});
    auto gau_c = torch::tensor({
                                   1, 0, 1,
                                   1, 0, 0,
                                   0, 1, 0,
                                   0, 0, 1
                               },
                               torch::dtype(torch::kFloat32)).view({-1, 3});
    gau_c = (gau_c - 0.5) / 0.28209;
    auto gau_a = torch::tensor({1, 1, 1, 1}, torch::dtype(torch::kFloat32)).view({-1, 1});

    return GaussianData(gau_xyz, gau_rot, gau_s, gau_a, gau_c);
}

GaussianData loadPly(std::filesystem::path filePath) {
    std::ifstream plyFile(filePath);
    if (!plyFile.is_open()) {
        std::cerr << "Failed to open file" << std::endl;
        return {};
    }

    // Skip the header as we already know the format
    std::string line;
    while (std::getline(plyFile, line)) {
        if (line == "end_header") {
            break;
        }
    }

    int numberOfVertices = 1026508; // As per your file's header
    std::vector<PlyVertex> vertices(numberOfVertices);

    for (int i = 0; i < numberOfVertices; ++i) {
        plyFile.read(reinterpret_cast<char*>(&vertices[i]), sizeof(PlyVertex));
    }

    plyFile.close();

    // Stack the tensors
    std::vector<float> x_values, y_values, z_values, opacity_values;
    for (const auto& v : vertices) {
        x_values.push_back(v.x);
        y_values.push_back(v.y);
        z_values.push_back(v.z);
        opacity_values.push_back(v.opacity);
    }

    auto x_tensor = torch::from_blob(x_values.data(), {static_cast<int64>(vertices.size())}, torch::kFloat32);
    auto y_tensor = torch::from_blob(y_values.data(), {static_cast<int64>(vertices.size())}, torch::kFloat32);
    auto z_tensor = torch::from_blob(z_values.data(), {static_cast<int64>(vertices.size())}, torch::kFloat32);
    auto xyz_tensor = torch::stack({x_tensor, y_tensor, z_tensor}, 1);
    auto opacities_tensor = torch::from_blob(opacity_values.data(), {static_cast<int64>(vertices.size())},
                                             torch::kFloat32);
    auto opacities_tensor_reshaped = opacities_tensor.unsqueeze(-1);

    // Continue reading the file based on header information

    int64_t num_vertices = vertices.size();
    auto features_dc = torch::zeros({num_vertices, 3, 1}, torch::kFloat32);
    std::vector<float> f_dc_0_values, f_dc_1_values, f_dc_2_values;
    for (const auto& v : vertices) {
        f_dc_0_values.push_back(v.f_dc[0]);
        f_dc_1_values.push_back(v.f_dc[1]);
        f_dc_2_values.push_back(v.f_dc[2]);
    }

    auto f_dc_0_tensor = torch::tensor(f_dc_0_values, torch::TensorOptions().dtype(torch::kFloat32)).unsqueeze(1);
    auto f_dc_1_tensor = torch::tensor(f_dc_1_values, torch::TensorOptions().dtype(torch::kFloat32)).unsqueeze(1);
    auto f_dc_2_tensor = torch::tensor(f_dc_2_values, torch::TensorOptions().dtype(torch::kFloat32)).unsqueeze(1);

    features_dc.index_put_({torch::indexing::Slice(), 0, torch::indexing::Slice()}, f_dc_0_tensor);
    features_dc.index_put_({torch::indexing::Slice(), 1, torch::indexing::Slice()}, f_dc_1_tensor);
    features_dc.index_put_({torch::indexing::Slice(), 2, torch::indexing::Slice()}, f_dc_2_tensor);


    std::vector<std::string> propertyNames; // Assume this is filled with your property names
    std::vector<std::string> scale_names; // Assume this is filled with your property names
    std::vector<std::string> rot_names; // Assume this is filled with your property names
    for (int i = 0; i < 45; ++i) {
        std::string str = "f_rest_" + std::to_string(i);
        propertyNames.push_back(str);
        scale_names.push_back("scale_" + std::to_string(i));
        rot_names.push_back("rot_" + std::to_string(i));
    }
    std::vector<std::string> extra_f_names;

    for (const auto& name : propertyNames) {
        if (name.rfind("f_rest_", 0) == 0) {
            // Check if the name starts with "f_rest_"
            extra_f_names.push_back(name);
        }
    }

    // Sort extra_f_names based on the integer suffix
    std::sort(extra_f_names.begin(), extra_f_names.end(), [](const std::string& a, const std::string& b) {
        int numA = std::stoi(a.substr(a.find_last_of('_') + 1));
        int numB = std::stoi(b.substr(b.find_last_of('_') + 1));
        return numA < numB;
    });

    int max_sh_degree = 3;
    int expected_count = 3 * std::pow(max_sh_degree + 1, 2) - 3;
    if (extra_f_names.size() != expected_count) {
        std::cerr << "Unexpected number of properties." << std::endl;
        return {}; // Or handle the error as needed
    }

    auto features_extra = torch::zeros({num_vertices, static_cast<int64_t>(extra_f_names.size())}, torch::kFloat32);

    for (int64_t idx = 0; idx < extra_f_names.size(); ++idx) {
        std::vector<float> property_values;
        for (const auto& v : vertices) {
            // Assuming 'getProperty' is a function to get the property value by name
            property_values.push_back(v.f_rest[idx]);
        }
        auto property_tensor = torch::from_blob(property_values.data(), {num_vertices}, torch::kFloat32);
        features_extra.index_put_({torch::indexing::Slice(), idx}, property_tensor);
    }

    features_extra = features_extra.view({num_vertices, 3, static_cast<int64_t>(std::pow(max_sh_degree + 1, 2) - 1)});
    features_extra = features_extra.transpose(1, 2);

    auto scales = torch::zeros({num_vertices, static_cast<int64_t>(scale_names.size())}, torch::kFloat32);

    for (int64_t idx = 0; idx < scale_names.size(); ++idx) {
        std::vector<float> scale_values;
        for (const auto& v : vertices) {
            scale_values.push_back(v.scale[idx]);
        }
        auto scale_tensor = torch::from_blob(scale_values.data(), {num_vertices}, torch::kFloat32);
        scales.index_put_({torch::indexing::Slice(), idx}, scale_tensor);
    }
    auto rotations = torch::zeros({num_vertices, static_cast<int64_t>(rot_names.size())}, torch::kFloat32);

    for (int64_t idx = 0; idx < rot_names.size(); ++idx) {
        std::vector<float> rot_vals;
        for (const auto& v : vertices) {
            rot_vals.push_back(v.rot[idx]);
        }
        auto rot_tensor = torch::from_blob(rot_vals.data(), {num_vertices}, torch::kFloat32);
        rotations.index_put_({torch::indexing::Slice(), idx}, rot_tensor);
    }

    auto rots_norm = rotations.norm(2, -1, true);
    rotations = rotations.div(rots_norm);
    scales = scales.exp();
    auto opacities = torch::sigmoid(opacities_tensor_reshaped);
    auto features_dc_reshaped = features_dc.view({-1, 3});

    // features_extra is already [1026508, 15, 3]

    auto features_dc_expanded = features_dc_reshaped.unsqueeze(-1).expand({-1, -1, 3});

    auto shs = torch::cat({features_dc_expanded, features_extra}, 1);
    return {xyz_tensor, rotations, scales, opacities, shs};
}

CudaImplementation::CudaImplementation(const RasterSettings* settings, std::vector<void*> handles) {
    this->handles = handles;

    cudaExtMem.resize(handles.size());
    cudaMemPtr.resize(handles.size());
    cudaMipMappedArrays.resize(handles.size());

    for (size_t i = 0; i < handles.size(); ++i) {
        hipExternalMemoryHandleDesc cudaExtMemHandleDesc{};
        memset(&cudaExtMemHandleDesc, 0, sizeof(cudaExtMemHandleDesc));
        cudaExtMemHandleDesc.size = settings->imageHeight * settings->imageWidth * 4;
        cudaExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
        cudaExtMemHandleDesc.handle.win32.handle = handles[i];
        cudaExtMemHandleDesc.flags = 0;
        checkCudaErrors(hipImportExternalMemory(&cudaExtMem[i], &cudaExtMemHandleDesc));

        // Step 3: CUDA memory copy
        /*
                hipExternalMemoryBufferDesc bufferDesc{};
                memset(&bufferDesc, 0, sizeof(bufferDesc));
                bufferDesc.offset = 0;
                bufferDesc.size = cudaExtMemHandleDesc.size;
                checkCudaErrors(hipExternalMemoryGetMappedBuffer(&cudaMemPtr[i], cudaExtMem[i], &bufferDesc));
                */
        cudaExternalMemoryMipmappedArrayDesc desc = {};
        memset(&desc, 0, sizeof(desc));

        hipChannelFormatDesc formatDesc;
        memset(&formatDesc, 0, sizeof(formatDesc));
        formatDesc.x = 8;
        formatDesc.y = 8;
        formatDesc.z = 8;
        formatDesc.w = 8;
        formatDesc.f = hipChannelFormatKindUnsigned;

        hipExtent extent = {0, 0, 0};
        extent.width = settings->imageWidth;
        extent.height = settings->imageHeight;
        extent.depth = 0;

        unsigned int flags = 0;
        flags |= hipArrayLayered;
        flags |= cudaArrayColorAttachment;

        desc.offset = 0;
        desc.formatDesc = formatDesc;
        desc.extent = extent;
        desc.flags = 0;
        desc.numLevels = 1;

        checkCudaErrors(cudaExternalMemoryGetMappedMipmappedArray(&cudaMipMappedArrays[i], cudaExtMem[i], &desc));
    }


    torch::Device device(torch::kCUDA);

    //viewmatrix = ConvertGlmMat4ToTensor(settings->viewMat).to(device);
    //projmatrix = ConvertGlmMat4ToTensor(settings->projMat).to(device);
    campos = ConvertGlmVec3ToTensor(settings->camPos).to(device);
    bg = torch::tensor({0.0, 0.0, 0.0}, torch::dtype(torch::kFloat32)).to(device);
    // Other parameters
    scale_modifier = settings->scaleModifier;
    tan_fovx = settings->tanFovX;
    tan_fovy = settings->tanFovY;
    image_height = settings->imageHeight;
    image_width = settings->imageWidth;
    degree = settings->shDegree;
    prefiltered = settings->prefilter;
    debug = settings->debug;

    //auto gaussianData = loadPly("C:\\Users\\mgjer\\Downloads\\models\\room\\point_cloud\\iteration_7000\\point_cloud.ply");
    auto gaussianData = naive_gaussian();
    // Example usage
    means3D = gaussianData.xyz.to(device);
    shs = gaussianData.sh.to(device);
    opacity = gaussianData.opacity.to(device);
    scales = gaussianData.scale.to(device);
    rotations = gaussianData.rot.to(device);
    cov3D_precomp = torch::tensor({}).to(device);
    colors = torch::tensor({}).to(device);
    degree = 0;

    //printTensor(means3D);
    //printTensor(rotations);
}

void CudaImplementation::updateGaussianData() {
    /*
    auto gaussianData = naive_gaussian();
    torch::Device device(torch::kCUDA);

    means3D = gaussianData.xyz.to(device);
    shs = gaussianData.sh.to(device);
    opacity = gaussianData.opacity.to(device);
    scales = gaussianData.scale.to(device);
    rotations = gaussianData.rot.to(device);
    cov3D_precomp = torch::tensor({}).to(device);
    colors = torch::tensor({}).to(device);

    degree = static_cast<int>(std::round(std::sqrt(gaussianData.sh_dim()))) - 1;
    */
}

void CudaImplementation::updateCameraPose(glm::mat4 view, glm::mat4 proj, glm::vec3 pos) {
    torch::Device device(torch::kCUDA);

    auto cameraPos = glm::vec3(glm::inverse(view)[3]);
    // Inverting the first and third rows of the view matrix
    // Note: GLM is column-major, so we access columns via view[col][row]
    view = glm::lookAt(-cameraPos, glm::vec3(0.0f), glm::vec3(0.0f, 0.0f, 1.0f));
    view[0][0] = -view[0][0];
    view[1][0] = -view[1][0];
    view[2][0] = -view[2][0];
    view[3][0] = -view[3][0];
    ////
    view[0][2] = -view[0][2];
    view[1][2] = -view[1][2];
    view[2][2] = -view[2][2];
    view[3][2] = -view[3][2];

    proj = glm::perspective(60.0f, 1024.0f / 1024.0f, 0.01f, 100.0f);
    glm::mat4 projView = proj * view;
    viewmatrix = ConvertGlmMat4ToTensor(view).to(device);
    projmatrix = ConvertGlmMat4ToTensor(projView).to(device);
    campos = ConvertGlmVec3ToTensor(cameraPos).to(torch::kFloat).to(device);
}

void CudaImplementation::updateSettings(const CudaImplementation::RasterSettings& settings) {
    scale_modifier = settings.scaleModifier;
}

void CudaImplementation::updateCameraIntrinsics(float hfox, float hfovy) {
    tan_fovx = hfox;
    tan_fovy = hfovy;
}


void CudaImplementation::draw(uint32_t i) {
    int rendered;
    torch::Tensor out_color, radii, geomBuffer, binningBuffer, imgBuffer;
    // Call the function
    std::tie(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer) = RasterizeGaussiansCUDA(
        bg, means3D, colors, opacity, scales, rotations,
        scale_modifier, cov3D_precomp, viewmatrix, projmatrix, tan_fovx, tan_fovy,
        image_height, image_width, shs, degree, campos, prefiltered, debug
    );
    // Ensure the tensor is on the CPU and is a byte tensor

    auto img = out_color.permute({1, 2, 0}); // Change [Channels, Height, Width] to [Height, Width, Channels]
    img = img.contiguous();
    auto alpha_channel = torch::ones({img.size(0), img.size(1), 1}, img.options());
    auto img_with_alpha = torch::cat({img, alpha_channel}, 2);
    img_with_alpha = img_with_alpha.contiguous();
    size_t data_size = img_with_alpha.numel(); // Assuming the tensor is of type torch::kFloat
    //printTensor(img_with_alpha);

    hipArray_t levelArray;
    checkCudaErrors(hipGetMipmappedArrayLevel(&levelArray, cudaMipMappedArrays[i], 0)); // 0 for the first level

    hipMemcpy3DParms p = {0};
    p.srcPtr = make_hipPitchedPtr(img_with_alpha.data_ptr(), 1024 * 16, 1024, 1024);
    p.dstArray = levelArray;
    p.extent = make_hipExtent(1024, 1024, 1); // depth is 1 for 2D
    p.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&p));
    /*
    hipMemcpy3DParms p = {0};
    p.srcPtr   = make_hipPitchedPtr(img_with_alpha.data_ptr(), 1024 * sizeof(float), 1024, 1024);
    p.dstArray = levelArray;
    p.extent   = make_hipExtent(width, height, depth); // depth is 1 for 2D
    p.kind     = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&p));
    */

    try {
        if (img.device().is_cuda()) {
            img = img.to(torch::kCPU);
            // Make sure the tensor is contiguous and in the format [Height, Width, Channels]
            img = img.contiguous();

            cv::Mat mat(img.size(0), img.size(1), CV_32FC(img.size(2)), img.data_ptr<float>());

            // Display the image
            cv::imshow("Output Image", mat);
            cv::waitKey(1); // Wait for a key press (use 0 for infinite wait)
        }
    }
    catch (const torch::Error& e) {
        std::cerr << "Error during tensor device check or transfer: " << e.what() << std::endl;
    }


    //checkCudaErrors(hipMemcpy(cudaMipMappedArrays[i],img_with_alpha.data_ptr() , data_size, hipMemcpyDeviceToDevice));
}
